#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <cuda_device_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_curand_states(hiprandState *states, unsigned long seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void initialize_values(float *array, hiprandState *states, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    hiprandState local = states[idx];
    array[idx] = hiprand_normal(&local) * 0.01f;
    states[idx] = local;
  }
}
